
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

using namespace std;

const int FILTER_WIDTH = 3;

//We will only use this filter in part 1
int FILTER[FILTER_WIDTH*FILTER_WIDTH] = {
    0, -1, 0, 
    -1, 5, -1, 
    0, -1, 0
};

// Display the first and last 10 items
// For debug only
void displayResult(const int original[], const int result[], int size) {
	cout << "Display result: ";

	for (int i = 0; i < 10; i++) {
		cout << result[i] << "\n";
	}
	cout << ".\n.\n.\n";

	for (int i = size - 10; i < size; i++) {
		cout << result[i] << "\n";
	}
}

void initData(string file, int **data, int *sizeX, int *sizeY) {
	int x;
	int y;
	long long i = 0;
	cout << "Reading "<< file << "... \n";
	ifstream myfile(file);
	if (myfile.is_open()) {
		myfile >> x;
		myfile >> y;

		int *temp = new int[x * y];
		for( i=0; i < x * y; i++){
			myfile >> temp[(int)i];
		}
		myfile.close();
		*data = temp;
		*sizeX = x;
		*sizeY = y;
	}
	else {
		cout << "ERROR: File " << file << " not found!\n";
		exit(0);
	}
	cout << i << " entries imported\n";
}

// Don't change this code
// We will evaluate your correctness based on the saved result, not printed output
void saveResult(string file, int data[], int sizeX, int sizeY) {
	long long i = 0;
	cout << "Saving data to "<< file <<"... \n";
	ofstream myfile(file, std::ofstream::out);
	if (myfile.is_open()) {
		myfile << sizeX << "\n";
		myfile << sizeY << "\n";
		for (i = 0; i < sizeX * sizeY; i++){
			myfile << data[i] << "\n";
		}
		myfile.close();
	}
	else {
		cout << "ERROR: Cannot save to " << file << "!\n";
		exit(0);
	}
	cout << i << " entries saved\n";
}

//TODO: Implement the kernel function
__global__ 
void sharpen(int *data, int *result, int sizeX, int sizeY) {
		int FILTER[9] = {0, -1, 0, -1, 5, -1, 0, -1, 0};
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < sizeX && y < sizeY) {
        int sum = 0;
        for (int fy = -1; fy <= 1; fy++) {
            for (int fx = -1; fx <= 1; fx++) {
                int ix = x + fx;
                int iy = y + fy;
                if (ix >= 0 && iy >= 0 && ix < sizeX && iy < sizeY) {
                    int fIdx = (fy + 1) * FILTER_WIDTH + (fx + 1);
                    int iIdx = iy * sizeX + ix;
                    sum += FILTER[fIdx] * data[iIdx];
                }
            }
        }
        result[y * sizeX + x] = sum;
    }
}

// GPU implementation
void GPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image

	// TODO: malloc memory, copy input
 	int *d_data, *d_result;
  int size = sizeX * sizeY * sizeof(int);
  hipMalloc((void **)&d_data, size);
  hipMalloc((void **)&d_result, size);

  // Copy the input data from the CPU to the GPU
	hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

	// Compute the number of blocks needed for the kernel launch
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks((sizeX + threadsPerBlock.x - 1) / threadsPerBlock.x, 
               (sizeY + threadsPerBlock.y - 1) / threadsPerBlock.y);	
	// Start timer for kernel
	// Don't change this function
	auto startKernel = chrono::steady_clock::now();
	
	// TODO: call the kernel function
  sharpen<<<numBlocks, threadsPerBlock>>>(d_data, d_result, sizeX, sizeY);
	// End timer for kernel and display kernel time
	hipDeviceSynchronize(); // <- DO NOT REMOVE
	auto endKernel = chrono::steady_clock::now();
	cout << "Kernel Elapsed time: " << chrono::duration <double, milli>(endKernel - startKernel).count() << "ms\n";

	// TODO: copy reuslt from device to host
	// Copy the result from the GPU to the CPU
  hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

	// TODO: free device memory <- important, keep your code clean
	// Free the memory allocated on the GPU
	hipFree(d_data);
	hipFree(d_result);
}


// CPU implementation
void CPU_Test(int data[], int result[], int sizeX, int sizeY) {
	// input:
	//	int data[] - int array holding the flattened original image
	//	int sizeX - the width of the image
	//	int sizeY - the height of the image
	// output:
	//	int result[] - int array holding the output image

	// TODO: sharpen the image with filter
	//       apply zero padding for the border

	for (int y = 0; y < sizeY; y++) {
        for (int x = 0; x < sizeX; x++) {
            int sum = 0;
            for (int fy = -1; fy <= 1; fy++) {
                for (int fx = -1; fx <= 1; fx++) {
                    int ix = x + fx;
                    int iy = y + fy;
                    if (ix >= 0 && iy >= 0 && ix < sizeX && iy < sizeY) {
                        int fIdx = (fy + 1) * FILTER_WIDTH + (fx + 1);
                        int iIdx = iy * sizeX + ix;
                        sum += FILTER[fIdx] * data[iIdx];
                    }
                }
            }
            result[y * sizeX + x] = sum;
        }
    }
}

// The input is a 2D grayscale image
// The image is flattened into a text file of pixel values.
int main(int argc, char *argv[]) {
	string inputFile = (argc == 1) ? "image_grey.txt" : argv[1];

	int sizeX;
	int sizeY;
	int *dataForCPUTest;
	int *dataForGPUTest;	

	initData(inputFile, &dataForCPUTest, &sizeX, &sizeY);
	initData(inputFile, &dataForGPUTest, &sizeX, &sizeY);

	int size = sizeX * sizeY;
	int *resultForCPUTest = new int[size];
	int *resultForGPUTest = new int[size];

	cout << "\n";

	cout << "CPU Implementation\n";

	auto startCPU = chrono::steady_clock::now();
	CPU_Test(dataForCPUTest, resultForCPUTest, sizeX, sizeY);
	auto endCPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endCPU - startCPU).count() << "ms\n";
	// For debug
	// displayResult(dataForCPUTest, resultForCPUTest, size);

	saveResult("grey_result_CPU.txt",resultForCPUTest, sizeX, sizeY);

	cout << "\n";
	cout << "GPU Implementation\n";

	auto startGPU = chrono::steady_clock::now();
	GPU_Test(dataForGPUTest, resultForGPUTest, sizeX, sizeY);
	auto endGPU = chrono::steady_clock::now();

	cout << "Elapsed time: " << chrono::duration <double, milli>(endGPU - startGPU).count() << "ms\n";

	// For debug
	// displayResult(dataForGPUTest, resultForGPUTest, size);
	saveResult("grey_result_GPU.txt",resultForGPUTest, sizeX, sizeY);

	return 0;
}
